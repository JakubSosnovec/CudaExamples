
#include <hip/hip_runtime.h>
#include <iostream>

int main(void) {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  std::cout << "CC: " << deviceProp.major << "." << deviceProp.minor << "\n";

  return 0;
}
